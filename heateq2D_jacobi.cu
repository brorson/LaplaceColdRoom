#include "hip/hip_runtime.h"
// This implements solution of the Laplace equation using
// finite differences.  The domain is a rectangle and I use
// Dirichlet BCs on the walls.  The BCs are those of the "cold
// room" example presented in the Northeastern class.
// The solution is found via the "relaxation method" which is
// the same as the Jacobi method for solving linear systems.
//
// SDB 5.29.2022
#include "heateq2D_jacobi.h"

//--------------------------------------------------------
// Error checking wrapper around CUDA fcns.  Copied from
// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess) {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}


//--------------------------------------------------------
// This runs on the device and implements the actual relaxation calculation.
__device__ void jacobi_step( const int tid, const float *du, float *du1) {
  // This performs one step of the Jacobi iteration on each
  // cuda core.  It is called from jacobi(), which also runs on the device.
  // Inputs:
  // tid = thread ID.
  // du = old computed temp profile.
  // Outputs:
  // du1 = new computed temp profile computed here.
  // c = scratchpad where square of (du-du1) is computed to judge convergence.
  int Nr = NR;
  int Nc = NC;
  
  // Figure out which matrix element to compute based on my
  // block and thread index values.
  int i = (int) tid/Nc;
  int j = (int) tid%Nc;
  
  // Do Jacobi step
  if ( (i>0) && (j>0) && (i<(Nr-1)) && (j<(Nc-1)) ) {
    // Only update nodes inside matrix, not on the boundary.
    du1[tid] = (du[LINDEX(Nr, Nc, i+1, j)]
		+ du[LINDEX(Nr, Nc, i-1, j)]
		+ du[LINDEX(Nr, Nc, i, j+1)]
		+ du[LINDEX(Nr, Nc, i, j-1)])/4.0f;
  } else {
    // This is a boundary node.  Just copy over the input.
    du1[tid] = du[tid];
  }
}

//--------------------------------------------------------
// This runs on the device.  It manages the Jacobi steps
// and computes part of the convergence criterion.
__global__ void jacobi(float *du, float *du1, float *dc_glob) {
  __shared__ float dc_loc[NTHD];  
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int ltid = threadIdx.x;   // my local index on this block.
  int bid = blockIdx.x;

  //printf("tid = %d, ltid = %d, bid = %d, NTHD = %d\n", tid, ltid, bid, NTHD);
  
  // Do Jacobi step at each point.
  jacobi_step(tid, du, du1);
    
  // synchronize threads
  __syncthreads();

  // Compute square of diff -- part of judging convergence.
  dc_loc[ltid] = (du1[tid]-du[tid])*(du1[tid]-du[tid]);

  // synchronize threads
  __syncthreads();

  // This does the reduction operation to compute the summed difference
  // between Jacobi steps.  This code handles the local reduction on this
  // block.
  int k = NTHD/2;
  while (k != 0){
    if (ltid < k) {
      dc_loc[ltid] += dc_loc[ltid + k];
      // printf("tid = %d, bid = %d, ltid = %d, k = %d, ltid+k = %d, dc_loc[ltid] = %e\n", tid, bid, ltid, k, ltid+k, dc_loc[ltid]);

    }
    __syncthreads();  // Wait for everybody to catch up.
    k /= 2;
  }

  // Now copy local result to correct place in c_glob.
  if (ltid == 0) {
    dc_glob[bid] = dc_loc[0];
    // printf("tid = %d, ltid = %d, bid = %d, dc_loc[0] = %e, dc_glob[bid] = %e\n",	   tid, ltid, bid, dc_loc[0], dc_glob[bid]);
  }

  //printf("tid = %d, dc_glob[tid] = %e\n", tid, dc_glob[tid]);
  
  // Copy updated temp matrix to old one in prep for next iteration.
  du[tid] = du1[tid];
}


//---------------------------------------------------
// Host-side convenience fcn
void linspace(float x0, float x1, int Npts, float *v) {
  // Returns vector v with Npts values from x0 to x1
  int i;
  float dx;
  dx = (x1-x0)/(Npts-1);
  for (i = 0; i < Npts; i++) {
    v[i] = x0 + i*dx;
  }
}

//-----------------------------------------------------
void print_matrix(const float* A, int m, int n) {
  // prints matrix as 2-dimensional table -- this is how we
  // usually think of matrices.
   int i, j;
   for (i = 0; i < m; i++) {
      for (j = 0; j < n; j++) {
          printf("%4.2e ", MATRIX_ELEMENT(A, m, n, i, j) );
      }
      printf("\n");
   }
}



//===================================================================
int main (void) {
  // First set up the physical parameters of the problem
  int Nr = NR;  // No of row points to sample.  Includes boundaries.
  int Nc = NC;  // No of col points to sample

  float Lx = 7.0f;   // X size of room.  Includes boundaries
  float Ly = 5.0f;   // Y size of room
  float LD = 2.0f;   // width of door at (x,y) = (Lx/2, 0)

  float Twall = 10.0f;  // Temp of wall
  float Tdoor = 50.0f;  // Temp of door

  // Create convenience vectors x and y.
  // I don't actually need them that much, but memory is cheap.
  float x[NR];
  float y[NC];
  linspace(-Lx/2.0f, Lx/2.0f, Nr, x);
  linspace(-Ly/2.0f, Ly/2.0f, Nc, y);  
  
  // Set up the temperature matrix on the host.
  // Host side is used for receiving the result and plotting.
  float *u;     // host
  u = (float *) malloc(Nr*Nc*sizeof(float));

  // Fill in temperature matrix u.
  for (int i=0; i<Nr*Nc; i++) {
    u[i] = Twall;
  }
  
  // Now set up BCs on right wall.  Use for loop to make temp either
  // Twall or Tdoor depending upon y value
  for (int i=0; i<NC; i++) {
    if (y[i] <= -LD/2 || y[i] >= LD/2) {
      // Outside door area
      u[LINDEX(Nr, Nc, i, Nc-1)] = Twall;
    } else {
      // Inside door area
      u[LINDEX(Nr, Nc, i, Nc-1)] = Tdoor;
    }
  }

  // Debug print
  //printf("Before computation, u = \n");
  //print_matrix(u, Nr, Nc);

  // Set up temperature matrices on device side
  // Device side is where the result is computed and placed into du
  float *du;     // pointer to var on device
  gpuErrchk( hipMalloc((void**)&du, NR*NC*sizeof(float)) );
  gpuErrchk( hipMemcpy(du, u, NR*NC*sizeof(float),
			hipMemcpyHostToDevice));

  float *du1;     // pointer to var on device
  gpuErrchk( hipMalloc((void**)&du1, NR*NC*sizeof(float)) );

  // variable where we will put the convergence diff collected by each block
  float *c_glob;   // host
  c_glob = (float *)malloc( NBLK *sizeof(float));
  float *dc_glob;  // device
  gpuErrchk( hipMalloc((void**)&dc_glob, NBLK*sizeof(float)) );  


  // Iterate over Jacobi epochs.
  int cnt;
  for (cnt=0; cnt<MAX_ITR; cnt++) {
    printf("------------------------------------------\n");
    printf("Jacobi epoch, cnt = %d, NBLK = %d, NTHD = %d\n", cnt, NBLK, NTHD);
  
    jacobi<<<NBLK,NTHD>>>(du, du1, dc_glob);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    // Copy c_glob back to the host and compute the sum-of-squares
    // difference between du and du1.
    //printf("Copy dc_glob back to host \n");    
    gpuErrchk( hipMemcpy(c_glob, &(dc_glob[0]), NBLK*sizeof(float),
			hipMemcpyDeviceToHost) );

    
    // Check for convergence
    printf("Compute sum-squared on host ... ");    
    float s = 0.0f;
    for (int i=0; i<NBLK; i++) {
      //printf("c_glob[%d] = %e\n", i, c_glob[i]);
      s += c_glob[i];
    }
    printf("s = %e\n", s);
    
    printf("Check for convergence on host ... ");        
    if (fabs(s) < TOL) {
      // Converged
      printf("converged!\n");
      break;
    } else {
      printf("not converged.  Loop again.\n");
    }
  }
  if (cnt == MAX_ITR) {
    // We failed to converge.  Error out.
    printf("====> Failed to converge after %d iterations.  Exiting...\n", MAX_ITR);
    return(-1);
  }
    
  // If we get here it's because we have converged. 
  // Copy the array 'du' back from the gpu to the cpu
  gpuErrchk( hipMemcpy(u, du, NR*NC*sizeof(float),
			hipMemcpyDeviceToHost) );

  // Debug print out of returned matrix.
  //printf("After computation, u = \n");
  //print_matrix(u, Nr, Nc);

  // Use VTK to make surface plot of result
  make_vtk_plot(Nc, Nr, y, x, u);
  
  //-----------------------------------------------------
  // Put temperature results into bitmap for display
  // Set up the host-side plotting stuff
  // This is old stuff I don't use any more.
  /*
  CPUBitmap bitmap( NR, NC );  // Bitmap on host to display results
  uint8_t *ptr = bitmap.get_ptr();
  for (int i=0; i<NR*NC; i++) {
    ptr[i*4 + 0] = (uint8_t) 4*u[i];
    ptr[i*4 + 1] = (uint8_t) 4*u[i];
    ptr[i*4 + 2] = (uint8_t) 4*u[i];
    ptr[i*4 + 3] = 255;
  }

  bitmap.display_and_exit();
  */

  // Clean up and exit.
  gpuErrchk( hipFree(du) );
  gpuErrchk( hipFree(du1) );  


  
}

 